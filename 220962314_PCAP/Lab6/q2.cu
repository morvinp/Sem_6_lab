#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void selsort(int *da, int *dc, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;  // Ensure we don't go out of bounds

    int k = 0;
    for (int j = 0; j < n; j++) {
        if ((da[j] < da[i]) || (da[j] == da[i] && j < i))
            k++;
    }
    dc[k] = da[i];  // Store sorted value at correct position
}

int main() {
    int n;
    printf("Enter the length of the vector: ");
    scanf("%d", &n);

    int *a = (int*)malloc(n * sizeof(int));
    int *c = (int*)malloc(n * sizeof(int));
    int *da, *dc;

    hipMalloc((void**)&da, n * sizeof(int));
    hipMalloc((void**)&dc, n * sizeof(int));

    printf("Enter the elements: ");
    for (int i = 0; i < n; i++) {
        scanf("%d", &a[i]);
    }

    hipMemcpy(da, a, n * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    selsort<<<numBlocks, blockSize>>>(da, dc, n);

    hipMemcpy(c, dc, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Sorted array: ");
    for (int i = 0; i < n; i++) {
        printf("%d ", c[i]);
    }
    printf("\n");

    hipFree(da);
    hipFree(dc);
    free(a);
    free(c);

    return 0;
}
 
