#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel function to add two matrices (each row computed by one thread)
__global__ void matrixAddRow(int *A, int *B, int *C, int rows, int cols) {
    int row = threadIdx.x;
    
    // Check if row is valid
    if (row < rows) {
        // Process the entire row
        for (int col = 0; col < cols; col++) {
            C[row * cols + col] = A[row * cols + col] + B[row * cols + col];
        }
    }
}

int main() {
    int rows = 4;
    int cols = 5;
    
    int size = rows * cols * sizeof(int);
    
    // Host matrices
    int *h_A = (int*)malloc(size);
    int *h_B = (int*)malloc(size);
    int *h_C = (int*)malloc(size);
    
    // Initialize matrices
    for (int i = 0; i < rows * cols; i++) {
        h_A[i] = i + 1;
        h_B[i] = (i + 1) * 2;
    }
    
    // Print matrix A
    printf("Matrix A:\n");
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%d ", h_A[i * cols + j]);
        }
        printf("\n");
    }
    
    // Print matrix B
    printf("Matrix B:\n");
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%d ", h_B[i * cols + j]);
        }
        printf("\n");
    }
    
    // Device matrices
    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    
    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    
    // Launch kernel with one thread per row
    matrixAddRow<<<1, rows>>>(d_A, d_B, d_C, rows, cols);
    
    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    
    // Print result matrix
    printf("Result Matrix C (A + B):\n");
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%d ", h_C[i * cols + j]);
        }
        printf("\n");
    }
    
    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    
    return 0;
}