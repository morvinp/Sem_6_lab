#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 1024

__global__ void CUDACount(char *A, unsigned int *d_count) {
    int i = threadIdx.x;
    if (A[i] == 'a') {
        atomicAdd(d_count, 1);
    }
}

int main() {
    char A[N];
    char *d_A;
    unsigned int count = 0, result;
    unsigned int *d_count;

    printf("Enter a string: ");
    fgets(A, N, stdin);  // Safe way to read input

    int length = strlen(A);
    if (A[length - 1] == '\n') A[length - 1] = '\0'; // Remove newline character

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Allocate memory
    hipMalloc((void**)&d_A, length * sizeof(char));
    hipMalloc((void**)&d_count, sizeof(unsigned int));

    // Copy data to device
    hipMemcpy(d_A, A, length * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice);

    // Check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA ERROR: %s\n", hipGetErrorString(error));
    }

    // Launch kernel
    CUDACount<<<1, length>>>(d_A, d_count);

    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA ERROR: %s\n", hipGetErrorString(error));
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Copy result back to host
    hipMemcpy(&result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);

    printf("Total occurrences of 'a' = %u\n", result);
    printf("Time taken: %f ms\n", elapsedTime);

    // Free memory
    hipFree(d_A);
    hipFree(d_count);

    return 0;
}
