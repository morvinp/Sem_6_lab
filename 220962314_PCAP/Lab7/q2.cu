#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>

// CUDA kernel to copy a string in progressively smaller sizes
__global__ void copyStringProgressively(char* d_result, const char* d_input, int str_len, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Each thread copies progressively smaller versions of the string
    if (idx < n) {
        // Calculate the number of characters to copy for this index
        int length_to_copy = str_len - idx;
        
        // Calculate the starting index in the result array for this thread
        int offset = idx * str_len;

        // Copy only the first `length_to_copy` characters into the result array
        for (int i = 0; i < length_to_copy; ++i) {
            d_result[offset + i] = d_input[i];
        }
        
        // Ensure the next part is null-terminated
        if (length_to_copy < str_len) {
            d_result[offset + length_to_copy        std::cout << std::endl;] = '\0';
        }
    }
}

int main() {
    std::string input_string = "PCAP";  // Example input string
    int n = 4;  // Number of progressively smaller versions of the string

    int str_len = input_string.length();
    int result_len = str_len * n;  // Total length of the result string

    // Allocate memory on the device
    char* d_input;
    char* d_result;
    hipMalloc((void**)&d_input, str_len * sizeof(char));
    hipMalloc((void**)&d_result, result_len * sizeof(char));

    // Copy input string to device memory
    hipMemcpy(d_input, input_string.c_str(), str_len * sizeof(char), hipMemcpyHostToDevice);

    // Define the number of threads and blocks
    int threadsPerBlock = 256;  // Adjust as needed
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock; // Ensure enough blocks to cover `n` copies

    // Launch kernel to copy progressively smaller strings
    copyStringProgressively<<<blocksPerGrid, threadsPerBlock>>>(d_result, d_input, str_len, n);

    // Allocate memory on the host for the result string
    char* h_result = new char[result_len + 1];  // +1 for null terminator

    // Copy the result back to host memory
    hipMemcpy(h_result, d_result, result_len * sizeof(char), hipMemcpyDeviceToHost);

    // Null-terminate the result string
    h_result[result_len] = '\0';

    // Output the result
    std::cout << "Result after progressively shortening the string: " << std::endl;
    for (int i = 0; i < n; ++i) {
        int length_to_print = str_len - i;
        for (int j = 0; j < length_to_print; ++j) {
            std::cout << h_result[i * str_len + j];
        }
    }

    // Free device memory
    hipFree(d_input);
    hipFree(d_result);

    // Free host memory
    delete[] h_result;

    return 0;
}
