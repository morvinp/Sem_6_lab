#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void transformMatrix(float *d_mat, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < M && col < N) {
        int index = row * N + col;
        float val = d_mat[index];
        for (int i = 1; i < row + 1; i++) { 
            d_mat[index] *= val;  // Multiply iteratively to raise power
        }
    }
}

int main() {
    int M, N;
    cout << "Enter matrix dimensions (M N): ";
    cin >> M >> N;

    float *h_mat = new float[M * N];

    cout << "Enter matrix elements row-wise:" << endl;
    for (int i = 0; i < M * N; i++) {
        cin >> h_mat[i];
    }

    float *d_mat;
    hipMalloc(&d_mat, M * N * sizeof(float));
    hipMemcpy(d_mat, h_mat, M * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((M + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    transformMatrix<<<numBlocks, threadsPerBlock>>>(d_mat, M, N);

    hipMemcpy(h_mat, d_mat, M * N * sizeof(float), hipMemcpyDeviceToHost);

    cout << "Modified matrix:" << endl;
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            cout << h_mat[i * N + j] << " ";
        }
        cout << endl;
    }

    hipFree(d_mat);
    delete[] h_mat;
    return 0;
}
